
#include <hip/hip_runtime.h>
#include <cstdio>
#include <numeric>

#define cudaCheckError()                                                       \
    {                                                                          \
        hipError_t e = hipGetLastError();                                    \
        if (e != hipSuccess)                                                  \
        {                                                                      \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__,           \
                   hipGetErrorString(e));                                     \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

// Computes the 1D index of a value in a contiguous buffer
// given its 2D coordinates and the width of a row.
//@@ #define IDX(row, col, width) ((???)*(???)+(???))  // FIXME
#define IDX(row, col, width) (row * width + col)

// computes c(i,j) = a(i,j) + b(i,j)
__global__ void add(int *a, int *b, int *c, int N, int M)
{
    //@@  int i=???;  // FIXME compute row the coordinates of the value
    //@@  int j=???;  // FIXME compute col the coordinates of the value
    //@@  if(??? && ???) {  // FIXME check boundaries
    //@@    int idx=IDX(i,j,M);  // keep this line
    //@@    c[idx] = a[idx] + b[idx];  // keep this line
    //@@  }
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M)
    {
        int idx = IDX(i, j, M);
        c[idx] = a[idx] + b[idx];
    }
}

int main()
{
    // 2048 rows and cols
    int N = 2 * 1024;
    int M = 2 * 1024;
    int *a, *b, *c;
    dim3 threads(32, 32);
    dim3 blocks(N / threads.x, M / threads.y);

    // Unified memory allocation
    hipMallocManaged(&a, N * M * sizeof(int));
    hipMallocManaged(&b, N * M * sizeof(int));
    hipMallocManaged(&c, N * M * sizeof(int));

    std::iota(a, a + N * M, 1);
    std::iota(b, b + N * M, 1);

    // Kernal launch
    add<<<blocks, threads>>>(a, b, c, N, M);
    hipDeviceSynchronize();
    cudaCheckError();

    // Check the results
    bool error = false;
    for (auto i = 0; i < N * M; i++)
    {
        if (error = a[i] + b[i] != c[i])
        {
            printf("ERROR at index %d.", i);
            break;
        }
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);
    cudaCheckError();

    if (!error)
    {
        printf("Test completed successfully.\n");
        return 0;
    }
    else
    {
        printf("WARNING there were some errors.\n");
        return 1;
    }
}
