
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define cudaCheckError() {                                                                       \
  hipError_t e=hipGetLastError();                                                        \
  if(e!=hipSuccess) {                                                                     \
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
      exit(EXIT_FAILURE);                                                                  \
  }                                                                                        \
}

// CUDA kernel to add elements of two arrays
__global__ void add(float *x, float *y)
{
  //@@ add a single element of x and y and store the result in y
}
 
int main(void)
{
  int N = 1<<20;  // 1M elements
  float *x, *y;
 
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
 
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
 
  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(x, y);
 
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  cudaCheckError();
 
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
  
  // Free memory
  hipFree(x);
  hipFree(y);
  cudaCheckError();

  if (maxError < 0.000001f) {
    printf("Test completed successfully.\n");
    return 0;
  } else {
    printf("WARNING there were some errors.\n");
    return 1;
  }
}