#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>

#include "kernels.h"

#define cudaCheckError()                                                       \
    {                                                                          \
        hipError_t e = hipGetLastError();                                    \
        if (e != hipSuccess)                                                  \
        {                                                                      \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__,           \
                   hipGetErrorString(e));                                     \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

int main()
{
    int cols = 2 * 1024;
    int rows = 2 * 1024;
    // int cols=4;  // Use less elements for debug if needed
    // int rows=4;
    float *d_buffer; //< Pointer to the 1D buffer we will manipulate

    //@@ Initialize grid and block sizes for later kernel launches.
    //@@ Use as many threads as possible.
    //@@ create 2D grid and blocks, you need to find the right structure to
    //store
    // those sizes
    //@@ ??? threads(???, ???);
    //@@ ??? blocks(???,
    //@@            ???);
    int bx = 32, by = 8;
    dim3 threads(by, by);
    dim3 blocks((cols + bx - 1) / bx, (rows + by - 1) / by);

    size_t pitch; //< we will store the pitch value in this variable

    // Allocate an 2D buffer with padding
    //@@ use hipMallocPitch to allocate this buffer
    //@@ hipMallocPitch(???);  // FIXME

    hipMallocPitch(&d_buffer, &pitch, cols * sizeof(float), rows);

    printf("Pitch d_buffer: %d\n", pitch);
    cudaCheckError();

    // The value we want our buffer to be filled with
    const float value = 5.f;

    // Initialize the buffer
    //@@ Call the fill2D kernel to fill d_buffer with `value`, see kernels.h for
    // the API
    //@@ fillZD<<<???, ???>>>(???);  // FIXME
    // Wait for GPU to finish and check for errors

    fill2D<<<blocks, threads>>>(d_buffer, value, cols, rows, pitch);

    hipDeviceSynchronize();
    cudaCheckError();

    // Check the content of the buffer on the device
    //@@ Call the check2D kernel to control device memory content, see kernels.h
    // for API
    //@@ check2D<<<?? ,???>>>(???);  // FIXME

    check2D<<<blocks, threads>>>(d_buffer, value, cols, rows, pitch);

    // Wait for GPU to finish and check for errors
    //@@ call CUDA device synchronisation function
    //@@ ???
    hipDeviceSynchronize();
    cudaCheckError();

    // Copy back buffer to host memory for inspection
    //@@ Allocate a buffer on the host
    //@@ float *h_x = (float*) std::malloc(???);  //FIXME
    //@@ Copy the buffer content from device to host
    //@@ use hipMemcpy2D
    //@@ hipMemcpy2D(???);  // FIXME
    float *h_x = (float*) std::malloc(cols * sizeof(float) * rows);
    hipMemcpy2D(h_x, cols * sizeof(float), d_x, pitch, cols * sizeof(float), rows, hipMemcpyDeviceToHost);

    cudaCheckError();

    // Check for errors
    float maxError = 0.0f;
    for (int i = 0; i < rows * cols; i++)
        maxError = std::fmax(maxError, std::fabs(host_buffer[i] - value));
    std::cout << "Max error: " << maxError << std::endl;
    bool noerror = (maxError < 0.0001f); // There is much smarter to do.

    // Clean up
    //@@ free d_buffer using CUDA primitives
    //@@ cuda???
    hipFree(d_buffer);
    cudaCheckError();

    std::free(host_buffer);

    // Useful return value
    if (noerror)
    {
        printf("Test completed successfully.\n");
        return 0;
    }
    else
    {
        printf("WARNING there were some errors.\n");
        return 1;
    }
}
